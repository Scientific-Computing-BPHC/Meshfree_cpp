#include "hip/hip_runtime.h"
#include "state_update_cuda.hpp"

__device__ inline void primitive_to_conserved(double* prim, double nx, double ny, double U[4], int idx);
__device__ inline void conserved_vector_Ubar(double* prim, double nx, double ny, double Mach, double gamma, double pr_inf, double rho_inf, double theta, double Ubar[4], int idx);

__global__ void call_func_delta_cuda(Point* globaldata, int numPoints, double cfl, dim3 thread_dim, int* connec, double* prim, double* prim_old)
{
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int idx = bx*thread_dim.x + tx;
    
    if(idx < numPoints)
	{
		double min_delt = 1.0;
		for(int i=0; i<20; i++)
		{
			int conn = connec[idx*20 + i];
			if (conn == 0) break;

            conn = conn -1; 

			double x_i = globaldata[idx].x;
			double y_i = globaldata[idx].y;
			double x_k = globaldata[conn].x;
			double y_k = globaldata[conn].y;

			double dist = hypot((x_k - x_i), (y_k - y_i));
			double mod_u = hypot(prim[conn*4 + 1], prim[conn*4 + 2]);
			double delta_t = dist/(mod_u + 3*sqrt(prim[conn*4 + 3]/prim[conn*4 + 0]));
			delta_t *= cfl;
			if (min_delt > delta_t)
				min_delt = delta_t;
		}
		globaldata[idx].delta = min_delt;
		for(int i=0; i<4; i++)
			prim_old[idx*4 + i] = prim[idx*4 + i];
	}
}

__global__ void state_update_cuda(Point* globaldata, int numPoints, Config configData, int iter, double res_old[1], int rk, int rks, \
    double* res_sqr, dim3 thread_dim, double* prim, double* prim_old, double* flux_res)
{
    int bx = blockIdx.x;
    int threadx = threadIdx.x;
    int idx = bx*thread_dim.x + threadx;
    
    double max_res = 0.0;

	double Mach = configData.core.mach;
	double gamma = configData.core.gamma;
	double pr_inf = configData.core.pr_inf;
	double rho_inf = configData.core.rho_inf;
	double theta = configData.core.aoa * (M_PI)/180.0;

    int euler = configData.core.euler;

    double U[4], Uold[4] = {0};

	if(idx < numPoints)
	{
		if(globaldata[idx].flag_1 == 0)
		{
			for(int i=0; i<4; i++)
			{
				U[i] = 0.0;
            }
			state_update_wall(globaldata, idx, max_res, res_sqr, U, Uold, rk, euler, prim, prim_old, flux_res);
		}
		else if(globaldata[idx].flag_1 == 2)
		{
			for(int i=0; i<4; i++)
			{
				U[i] = 0.0;
            }
			state_update_outer(globaldata, idx, Mach, gamma, pr_inf, rho_inf, theta, max_res, res_sqr, U, Uold, rk, euler, prim, prim_old, flux_res);
		}
		else if(globaldata[idx].flag_1 == 1)
		{
			for(int i=0; i<4; i++)
			{
				U[i] = 0.0;
            }
			state_update_interior(globaldata, idx, max_res, res_sqr, U, Uold, rk, euler, prim, prim_old, flux_res);
		}
    }
}

__device__ void state_update_wall(Point* globaldata, int idx, double max_res, double* res_sqr, double U[4], double Uold[4], int rk, int euler, double* prim, double* prim_old, double* flux_res)
{
    double nx = globaldata[idx].nx;
    double ny = globaldata[idx].ny;

    primitive_to_conserved(prim, nx, ny, U, idx);
    primitive_to_conserved(prim_old, nx, ny, Uold, idx);

    double temp = U[0];

    for (int iter=0; iter<4; iter++)
    {
        U[iter] = U[iter] - 0.5 * euler * flux_res[idx*4 + iter];
    }

    if (rk == 2)
    {
        for (int iter=0; iter<4; iter++)
            U[iter] = U[iter] * ((double)1.0)/3.0 + Uold[iter] * ((double)2.0)/3.0;
    }

    U[2] = 0.0;
    double U2_rot = U[1];
    double U3_rot = U[2];
    U[1] = U2_rot*ny + U3_rot*nx;
    U[2] = U3_rot*ny - U2_rot*nx;
    res_sqr[idx] = (U[0] - temp)*(U[0] - temp);

    Uold[0] = U[0];
    temp = 1.0 / U[0];
    Uold[1] = U[1]*temp;
    Uold[2] = U[2]*temp;
    Uold[3] = (0.4*U[3]) - ((0.2 * temp) * (U[1] * U[1] + U[2] * U[2]));
    for(int i=0; i<4; i++)
    {
    	prim[idx*4 + i] = Uold[i];
    }


	// if(idx ==0)
	// {
	// 	printf("\n");
	// 	for(int index = 0; index<4; index++)
	// 	{
	// 		printf("%.17f   ", prim[idx*4 + index]);
	// 	}
	// }

}

__device__ void state_update_outer(Point* globaldata, int idx, double Mach, double gamma, double pr_inf, double rho_inf, double theta, double max_res, double* res_sqr, \
    double U[4], double Uold[4], int rk, int euler, double* prim, double* prim_old, double* flux_res)
{
    double nx = globaldata[idx].nx;
    double ny = globaldata[idx].ny;

    conserved_vector_Ubar(prim, nx, ny, Mach, gamma, pr_inf, rho_inf, theta, U, idx);
    conserved_vector_Ubar(prim_old, nx, ny, Mach, gamma, pr_inf, rho_inf, theta, Uold, idx);

    double temp = U[0];
    for (int iter=0; iter<4; iter++)
    {
        U[iter] = U[iter] - 0.5 * euler * flux_res[idx*4 + iter];
    }
    if (rk == 2)
    {
        for (int iter=0; iter<4; iter++)
            U[iter] = U[iter] * ((double)1.0)/3.0 + Uold[iter] * ((double)2.0)/3.0;
    }

    double U2_rot = U[1];
    double U3_rot = U[2];
    U[1] = U2_rot*ny + U3_rot*nx;
    U[2] = U3_rot*ny - U2_rot*nx;
    res_sqr[idx] = (U[0] - temp)*(U[0] - temp);

    Uold[0] = U[0];
    temp = 1.0 / U[0];
    Uold[1] = U[1]*temp;
    Uold[2] = U[2]*temp;
    Uold[3] = (0.4*U[3]) - ((0.2 * temp) * (U[1] * U[1] + U[2] * U[2]));
    for(int i=0; i<4; i++)
    {
    	prim[idx*4 + i] = Uold[i];
    }

}

__device__ void state_update_interior(Point* globaldata, int idx, double max_res, double* res_sqr, double U[4], double Uold[4], int rk, int euler, double* prim, double* prim_old, \
    double* flux_res)
{
    double nx = globaldata[idx].nx;
    double ny = globaldata[idx].ny;

    primitive_to_conserved(prim, nx, ny, U, idx);
    primitive_to_conserved(prim_old, nx, ny, Uold, idx);

    double temp = U[0];
    for (int iter=0; iter<4; iter++)
        U[iter] = U[iter] - 0.5 * euler * flux_res[idx*4 + iter];
    if (rk == 2)
    {
        for (int iter=0; iter<4; iter++)
            U[iter] = U[iter] * ((double)1.0)/3.0 + Uold[iter] * ((double)2.0)/3.0;
    }

    double U2_rot = U[1];
    double U3_rot = U[2];
    U[1] = U2_rot*ny + U3_rot*nx;
    U[2] = U3_rot*ny - U2_rot*nx;
    res_sqr[idx] = (U[0] - temp)*(U[0] - temp);

    Uold[0] = U[0];
    temp = 1.0 / U[0];
    Uold[1] = U[1]*temp;
    Uold[2] = U[2]*temp;
    Uold[3] = (0.4*U[3]) - ((0.2 * temp) * (U[1] * U[1] + U[2] * U[2]));
    for(int i=0; i<4; i++)
    {
    	prim[idx*4 + i] = Uold[i];
    }

}

__device__ inline void primitive_to_conserved(double* prim, double nx, double ny, double U[4], int idx)
{
	double rho = prim[idx*4 + 0];
    U[0] = rho;
    double temp1 = rho * prim[idx*4 + 1];
    double temp2 = rho * prim[idx*4 + 2];
    U[1] = temp1*ny - temp2*nx;
    U[2] = temp1*nx + temp2*ny;
    U[3] = 2.5*prim[idx*4 + 3] + 0.5*(temp1*temp1 + temp2*temp2)/rho;
}

__device__ inline void conserved_vector_Ubar(double* prim, double nx, double ny, double Mach, double gamma, double pr_inf, double rho_inf, double theta, double Ubar[4], int idx)
{
	double u1_inf = Mach*cos(theta);
    double u2_inf = Mach*sin(theta);

    double tx = ny;
    double ty = -nx;

    double u1_inf_rot = u1_inf*tx + u2_inf*ty;
    double u2_inf_rot = u1_inf*nx + u2_inf*ny;

    double temp1 = (u1_inf_rot * u1_inf_rot + u2_inf_rot*u2_inf_rot);
    double e_inf = (pr_inf/(rho_inf*(gamma-1))) + 0.5 * (temp1);

    double beta = (0.5 * rho_inf)/pr_inf;
    double S2 = u2_inf_rot * sqrt(beta);
    double B2_inf = exp(-S2*S2)/(2.0*sqrt(M_PI*beta));
    double A2n_inf = 0.5 * (1 - erf(S2));

    double rho = prim[idx*4 + 0];
    double u1 = prim[idx*4 + 1];
    double u2 = prim[idx*4 + 2];
    double pr = prim[idx*4 + 3];

    double u1_rot = u1*tx + u2*ty;
    double u2_rot = u1*nx + u2*ny;

    temp1 = (u1_rot*u1_rot + u2_rot*u2_rot);
    double e = (pr/(rho*(gamma-1))) + 0.5*(temp1);

    beta = (rho)/(2.0*pr);
    S2 = u2_rot*sqrt(beta);
    double B2 = exp(-S2*S2)/(2.0*sqrt(M_PI*beta));
    double A2p = 0.5*(1.0 + erf(S2));

    Ubar[0] = (rho_inf*A2n_inf) + (rho*A2p);

    Ubar[1] = (rho_inf*u1_inf_rot*A2n_inf) + (rho*u1_rot*A2p);

    temp1 = rho_inf*(u2_inf_rot*A2n_inf - B2_inf);
    double temp2 = rho*(u2_rot*A2p + B2);
    Ubar[2] = (temp1 + temp2);

    temp1 = (rho_inf*A2n_inf* e_inf - 0.5*rho_inf*u2_inf_rot*B2_inf);
    temp2 = (rho*A2p*e + 0.5*rho*u2_rot*B2);

    Ubar[3] = (temp1 + temp2);
}

